#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <ops.cuh>
#include <kernels.cuh>
#include <cub/device/device_scan.cuh>
#include <limits>
#include <BinSearch.h>
#include <cassert>
#include <common.h>

#define ERR_NOT_IMPLEMENTED 100


using namespace BinSearch;
using std::cout;
using std::endl;

void histogramScatterAdd2D(float* histogram, int *index1, int *index2, float *src, int maxidx1, int n)
{
  int threads = 512;
  int num_blocks = n/threads;
  num_blocks = n % threads == 0 ? num_blocks : num_blocks + 1;
  kHistogramScatterAdd2D<<<num_blocks, 512>>>(histogram, index1, index2, src, maxidx1, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<num_blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int num_blocks = n/1024;
  num_blocks = n % 1024 == 0 ? num_blocks : num_blocks + 1;
  kQuantize<<<num_blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n, hipStream_t stream)
{
  int num_blocks = n/1024;
  num_blocks = n % 1024 == 0 ? num_blocks : num_blocks + 1;
  kDequantize<<<num_blocks, 1024, 0, stream>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC, int DATA_TYPE> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, int blocksize, const int n)
{
  int num_blocks = n/blocksize;
  num_blocks = n % blocksize == 0 ? num_blocks : num_blocks + 1;

  if(blocksize == 4096)
    kQuantizeBlockwise<T, 4096, 4, STOCHASTIC, DATA_TYPE><<<num_blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 2048)
    kQuantizeBlockwise<T, 2048, 4, 0, DATA_TYPE><<<num_blocks, 512>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 1024)
    kQuantizeBlockwise<T, 1024, 4, 0, DATA_TYPE><<<num_blocks, 256>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 512)
    kQuantizeBlockwise<T, 512, 2, 0, DATA_TYPE><<<num_blocks, 256>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 256)
    kQuantizeBlockwise<T, 256, 2, 0, DATA_TYPE><<<num_blocks, 128>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 128)
    kQuantizeBlockwise<T, 128, 2, 0, DATA_TYPE><<<num_blocks, 64>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 64)
    kQuantizeBlockwise<T, 64, 2, 0, DATA_TYPE><<<num_blocks, 32>>>(code, A, absmax, out, rand, rand_offset, n);


  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int DATA_TYPE> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n, hipStream_t stream)
{
  // printf("stream==%d\n",stream);
  int num_blocks = n/blocksize;
  num_blocks = n % blocksize == 0 ? num_blocks : num_blocks + 1;
  int tile_size = (DATA_TYPE > 0) ? 1024 : 512;
  if(DATA_TYPE > 0)
    kDequantizeBlockwise<T, 512, 64, 8, DATA_TYPE><<<(n+tile_size-1)/tile_size, 64, 0, stream>>>(code, A, absmax, out, blocksize/2, n);
  else
    kDequantizeBlockwise<T, 512, 64, 8, DATA_TYPE><<<(n+tile_size-1)/tile_size, 64, 0, stream>>>(code, A, absmax, out, blocksize, n);

  CUDA_CHECK_RETURN(hipPeekAtLastError());
}



template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p,
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float beta3, const float alpha, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, bool skip_zeros, const int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
    case ADEMAMIX:
      if(max_unorm > 0.0f)
			{
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<num_blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<num_blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, beta3, alpha, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:
      if(max_unorm > 0.0f)
			{
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<num_blocks, 512>>>(g, p, state1, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
    case LION:
      // in lion, the momentum update after the parameter update
      kOptimizer32bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());

      if(max_unorm > 0.0f)
      {
        CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<num_blocks, 512>>>(g, p, state1, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
      break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr,
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<num_blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<num_blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 256>>>(p, g, state1, unorm, beta1, beta2, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
    case LION:
      // in lion, the momentum update happens after the parameter update
      kOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
                                                            quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());

      CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
      kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 256>>>(p, g, state1, unorm, beta1, beta2, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
      break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 256
#define NUM_2STATE 1
#define BLOCKSIZE_1STATE 256
#define NUM_1STATE 1

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(
    T* p,
    T* g,
    unsigned char* state1,
    unsigned char* state2,
    float beta1,
    float beta2,
    float beta3,
    float alpha,
    float eps,
    int step,
    float lr,
    float* quantiles1,
    float* quantiles2,
    float* absmax1,
    float* absmax2,
    float weight_decay,
    const float gnorm_scale,
    bool skip_zeros,
    int n
) {

	int num_blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
    case ADEMAMIX:
			num_blocks = n/BLOCKSIZE_2STATE;
			num_blocks = n % BLOCKSIZE_2STATE == 0 ? num_blocks : num_blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<num_blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(
				p, g, state1, state2, beta1, beta2, beta3, alpha, eps, step, lr,
				quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale,
				skip_zeros, n
			);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
    case ADAGRAD:
    case LION:
			num_blocks = n/BLOCKSIZE_1STATE;
			num_blocks = n % BLOCKSIZE_1STATE == 0 ? num_blocks : num_blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<num_blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, skip_zeros, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int num_blocks = n/2048;
  num_blocks = n % 2048 == 0 ? num_blocks : num_blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<num_blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

			status = hipblasGemmEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, beta,
					C, HIP_R_32I, ldc,
          HIP_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

void strided_gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc,
                    long long int strideA, long long int strideB, long long int strideC, int batchCount)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

  //cout << transposeA << transposeB << endl;
  //printf("%i %i %i\n", m,n,k);
  //printf("%i %i %i\n", lda,ldb,ldc);
  //printf("%i %i %i\n", strideA, strideB, strideC);
  //printf("%i\n", batchCount);

			status = hipblasGemmStridedBatchedEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, (long long int)strideA, B, HIP_R_8I, ldb, (long long int)strideB, beta,
					C, HIP_R_32I, ldc, (long long int)strideC, batchCount,
          HIP_R_32I, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}


template<int ORDER> hipblasLtOrder_t get_order()
{
	switch(ORDER)
	{
		case ROW:
      return HIPBLASLT_ORDER_ROW;
			break;
    case COL:
      return HIPBLASLT_ORDER_COL;
      break;
    case COL32:
      return CUBLASLT_ORDER_COL32;
      break;
    case COL_TURING:
      return CUBLASLT_ORDER_COL4_4R2_8C;
      break;
    case COL_AMPERE:
      return CUBLASLT_ORDER_COL32_2R_4R4;
      break;
		default:
			break;
  }

	return HIPBLASLT_ORDER_ROW;
}

template hipblasLtOrder_t get_order<ROW>();
template hipblasLtOrder_t get_order<COL>();
template hipblasLtOrder_t get_order<COL32>();
template hipblasLtOrder_t get_order<COL_TURING>();
template hipblasLtOrder_t get_order<COL_AMPERE>();


template<int ORDER> int get_leading_dim(int dim1, int dim2)
{
	switch(ORDER)
	{
		case ROW:
      return dim2;
			break;
    case COL:
      return dim1;
      break;
    case COL32:
      // 32*row tiles
      return dim1*32;
      break;
    case COL_TURING:
      return 32*roundoff(dim1, 8);
      break;
    case COL_AMPERE:
      // 32*32 tiles
      return 32*roundoff(dim1, 32);
      break;
		default:
			return 0;
			break;
  }
}

template <typename T, int SRC, int TARGET, bool transpose, int DTYPE> void transform(hipblasLtHandle_t ltHandle, T *A, T *out, int dim1, int dim2)
{
  hipblasLtOrder_t orderA = get_order<SRC>();
  hipblasLtOrder_t orderOut = get_order<TARGET>();
  int ldA = get_leading_dim<SRC>(dim1, dim2);
  int ldOut = get_leading_dim<TARGET>(dim1, dim2);

  hipblasLtMatrixLayout_t A_desc = NULL, out_desc = NULL;
  hipblasLtMatrixTransformDesc_t A2Out_desc = NULL;
  hipblasOperation_t opTranspose = HIPBLAS_OP_T;
  float transformAlpha = 1.0f, transformBeta = 0.0f;


  if(DTYPE == 8)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_8I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_8I, dim1, dim2, ldOut));
  }
  else if(DTYPE == 32)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_32I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_32I, dim1, dim2, ldOut));
  }
  else
  {
    printf("ERROR WRONG TYPE FOR TRANSFORM: %i\n", DTYPE);
  }

  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(A_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderA, sizeof(orderA)));
  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(out_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderOut, sizeof(orderOut)));

  checkCublasStatus(hipblasLtMatrixTransformDescCreate(&A2Out_desc, HIP_R_32F));

  if(transpose){ checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(A2Out_desc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose))); }

  checkCublasStatus(hipblasLtMatrixTransform(ltHandle, A2Out_desc, &transformAlpha, A, A_desc, &transformBeta, NULL, NULL, out, out_desc, 0));

  if (A_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(A_desc));
  if (out_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(out_desc));
  if (A2Out_desc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(A2Out_desc));
}

template <int DTYPE_OUT, int SCALE_ROWS> int igemmlt(
  hipblasLtHandle_t ltHandle,
  int m, int n, int k,
  const int8_t * A,
  const int8_t * B,
  void * C,
  float * row_scale,
  int lda, int ldb, int ldc,
  hipStream_t stream
) {

  // Calculate C = A^T @ B, in col-major layout.
  //
  // Use the IMMA kernels requires:
  // * A must be transposed and B must be non-transposed.
  // * Dimensions m and k must be multiples of 4.
  // * All pointers must be 4-byte aligned; 16-byte alignment preferred.

  int has_error = 0;

  hipblasLtMatmulDesc_t matmulDesc;
  hipblasLtMatrixLayout_t aDesc, bDesc, cDesc;
  hipblasOperation_t opT = HIPBLAS_OP_T;

  hipDataType outType = DTYPE_OUT == 32 ? HIP_R_32I : HIP_R_8I;
  hipDataType scaleType = DTYPE_OUT == 32 ? HIP_R_32I : HIP_R_32F;

  hipblasLtPointerMode_t pointerMode = CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;

  has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_8I, m, k, lda));
  has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_8I, m, n, ldb));
  has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&cDesc, outType, k, n, ldc));

  // Default layout order is col major

  has_error |= checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, scaleType));
  has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opT, sizeof(opT)));

  if (DTYPE_OUT == 32) {
      int alpha = 1, beta = 0;
      has_error |= checkCublasStatus(hipblasLtMatmul(
        ltHandle, matmulDesc,
        &alpha, A, aDesc,
        B, bDesc, &beta,
        (int32_t*)C, cDesc,
        (int32_t*)C, cDesc,
        NULL, NULL, 0, stream
      ));
  } else {
    // This path is unlikely to be used, as 8-bit accumulation can lead to likely overflows.

    if (!SCALE_ROWS) {
      float alpha = 1.0f, beta = 0.0f;
      has_error |= checkCublasStatus(hipblasLtMatmul(
        ltHandle, matmulDesc,
        &alpha, A, aDesc,
        B, bDesc, &beta,
        (int8_t*)C, cDesc,
        (int8_t*)C, cDesc,
        NULL, NULL, 0, stream
      ));
    } else {
      hipblasLtPointerMode_t alphaVec = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
      float beta = 0.0f;
      has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(
        matmulDesc,
        HIPBLASLT_MATMUL_DESC_POINTER_MODE,
        &pointerMode,
        sizeof(alphaVec)
      ));
      has_error |= checkCublasStatus(hipblasLtMatmul(
        ltHandle, matmulDesc,
        row_scale, A, aDesc,
        B, bDesc, &beta,
        (int8_t*)C, cDesc,
        (int8_t*)C, cDesc,
        NULL, NULL, 0, stream
      ));
    }
  }

  has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(cDesc));
  has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(bDesc));
  has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(aDesc));
  has_error |= checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));

  if(has_error == 1)
    printf("error detected");

  return has_error;
}

int fill_up_to_nearest_multiple(int value, int multiple)
{
  return value + (value % multiple == 0 ? 0 : (multiple - (value % multiple)));
}

void dequant_mm_int32_fp16(int *A, float *rowStats, float *colStats, half *out, half *bias, int numRows, int numCols, hipStream_t stream)
{
  const int threads = 512;
  const int num_per_thread = 4;
  const int num_per_block = threads * num_per_thread;
  const int n = numRows*numCols;
  const int num_blocks = (n + num_per_block - 1) / num_per_block;

  kdequant_mm_int32_fp16<num_per_thread, threads><<<num_blocks, threads, 0, stream>>>(A, rowStats, colStats, out, bias, numRows, numCols, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void int8VectorQuant(half * __restrict__ A, int8_t *out, float *rowStats, float threshold, int rows, int cols, hipStream_t stream) {
  if (threshold == 0.0) {
    kInt8VectorQuant<half, 1024, 0><<<rows, 1024, 0, stream>>>(A, out, rowStats, threshold, rows, cols);
  } else {
    kInt8VectorQuant<half, 1024, 1><<<rows, 1024, 0, stream>>>(A, out, rowStats, threshold, rows, cols);
  }
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void getRowStats(half *A, float *rowStats, float threshold, int rows, int cols, hipStream_t stream) {
  if (threshold == 0.0)
    kgetRowStats<half, 1024, 0><<<rows, 1024, 0, stream>>>(A, rowStats, threshold, rows, cols);
  else
    kgetRowStats<half, 1024, 1><<<rows, 1024, 0, stream>>>(A, rowStats, threshold, rows, cols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <int FORMAT, int TRANSPOSE> void transformRowToFormat(char * A, char *out, int rows, int cols)
{
  int threads = 256;
  int items_per_thread = 8;
  // we load 128 column values per warp
  int tile_cols = 32*items_per_thread;
  int tile_rows = 32;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
	int row_tiles = (tiledRows/tile_rows);
	int col_tiles = (tiledCols/tile_cols);
	row_tiles = row_tiles > 0 ? row_tiles : 1;
	col_tiles = col_tiles > 0 ? col_tiles : 1;
  int num_blocks = row_tiles * col_tiles;

  int outCols = fill_up_to_nearest_multiple(cols, 32);
  int outRows = fill_up_to_nearest_multiple(rows, 32);
  if(FORMAT == COL_TURING)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 8);
    else
      outRows = fill_up_to_nearest_multiple(rows, 8);
  }
  else if(FORMAT == COL_AMPERE)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 32);
    else
      outRows = fill_up_to_nearest_multiple(rows, 32);
  }
  else
  {
    if(TRANSPOSE)
    {
      outCols = fill_up_to_nearest_multiple(rows, 32);
      outRows = cols;
    }
  }

  kTransformRowToFormat<256, 8, 32, 32*8, TRANSPOSE, FORMAT><<<num_blocks, threads>>>(A, out, rows, cols, tiledCols, outRows, outCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void spmm_coo(hipsparseHandle_t handle, int *A_rowidx, int *A_colidx, half *A_vals, int A_nnz, int A_rows, int A_cols, int B_cols, int ldb, half *B, int ldc, half* C, bool transposed_B)
{
    hipsparseSpMatDescr_t descA;
    hipsparseDnMatDescr_t descB, descC;

    float alpha = 1.0f;
    float beta = 0.0f;
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    CHECK_CUSPARSE( hipsparseCreateCoo(&descA, A_rows, A_cols, A_nnz,
                                      A_rowidx, A_colidx, A_vals,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) );
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&descC, A_rows, B_cols, ldc, C,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) );
    // Create dense matrix B
    if(transposed_B)
    {
      int tmp = A_cols;
      A_cols = B_cols;
      B_cols = tmp;
    }

    CHECK_CUSPARSE( hipsparseCreateDnMat(&descB, A_cols, B_cols, ldb, B,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) );
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 transposed_B ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descA, descB, &beta, descC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) );
    CUDA_CHECK_RETURN( hipMalloc(&dBuffer, bufferSize) );

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 transposed_B ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, descA, descB, &beta, descC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(descA) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(descB) );
    CHECK_CUSPARSE( hipsparseDestroyDnMat(descC) );
    CUDA_CHECK_RETURN( hipFree(dBuffer) );
}

template <typename T, int BITS> void spmm_coo_very_sparse_naive(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, T *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB)
{

  kspmm_coo_very_sparse_naive<T, 8, BITS><<<nnz_rows, 256>>>(max_count, max_idx, offset_rowidx, rowidx, colidx, values, B, out, dequant_stats, nnz, rowsA, rowsB, colsB);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}


template <int FORMAT> void extractOutliers(char * A, int *idx, char *out, int idx_size, int rows, int cols)
{
  int threads = 256;
  // we load 128 column values per warp
  int tiledCols = tiledCols = fill_up_to_nearest_multiple(cols, 32);
  int tiledRows = 0;

	int num_blocks = idx_size;

  if(FORMAT == COL_TURING)
  {
      tiledRows = fill_up_to_nearest_multiple(rows, 8);
  }
  else if(FORMAT == COL_AMPERE)
  {
      tiledRows = fill_up_to_nearest_multiple(rows, 32);
	}

  kExtractOutliers<FORMAT><<<num_blocks, threads>>>(A, idx, out, idx_size, rows, cols, tiledRows, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}




template <typename T> void gemm_host(int m, int n, int k, T * A,  T* B,  T * out,  int lda, int ldb, int ldc, int bits)
{

	int num_blocks = (m+31)/32;

  if(bits == 32)
    gemm_device<T, 32, 32><<< num_blocks, 32, 0, 0 >>>(m,  n,  k, A,  B,  out, lda, ldb, ldc);
  if(bits == 16)
    gemm_device<T, 16, 160><<< num_blocks, 160, 0, 0 >>>(m,  n,  k, A,  B,  out, lda, ldb, ldc);
}

template <typename T> void gemm_4bit_inference(int m, int n, int k, T * A,  unsigned char* B,  float *absmax, T * out,  int lda, int ldb, int ldc, int blocksize)
{

	int num_blocks = (m+31)/32;

  kgemm_4bit_inference<T, 96><<< num_blocks, 96, 0, 0 >>>(m,  n,  k, A,  B, absmax, out, lda, ldb, ldc, blocksize);
}

template <typename T, int BITS> void gemm_4bit_inference_naive(int m, int n, int k, T * A,  unsigned char* B,  float *absmax, float *datatype, T * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream)
{

	int num_blocks = (m+3)/4;
  kgemm_4bit_inference_naive<T, 128, BITS><<< num_blocks, 128, 0, stream>>>(m,  n,  k, A,  B, absmax, datatype, out, lda, ldb, ldc, blocksize);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int FUNC> void func(T *A, T *B, T value, long n)
{
  int threads = 512;
  int blocks = n/threads;
  blocks = n % threads == 0 ? blocks : blocks + 1;
  blocks = blocks > 65535 ? 65535 : blocks;
  kfunc<T, FUNC><<<blocks, 512>>>(A, B, value, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void func<float, FILL>(float *A, float *B, float value, long n);
template void func<unsigned char, FILL>(unsigned char *A, unsigned char *B, unsigned char value, long n);
template void func<float, ARANGE>(float *A, float *B, float value, long n);
template void func<float, _MUL>(float *A, float *B, float value, long n);

template void gemm_4bit_inference<half>(int m, int n, int k, half * A,  unsigned char* B,  float *absmax, half * out,  int lda, int ldb, int ldc, int blocksize);
template void gemm_4bit_inference_naive<half, 16>(int m, int n, int k, half * A,  unsigned char* B,  float *absmax, float *datatype, half * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream);
template void gemm_4bit_inference_naive<__hip_bfloat16, 16>(int m, int n, int k, __hip_bfloat16 * A,  unsigned char* B,  float *absmax, float *datatype, __hip_bfloat16 * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream);
template void gemm_4bit_inference_naive<float, 32>(int m, int n, int k, float * A,  unsigned char* B,  float *absmax, float *datatype, float * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream);

//template void gemm_host<float>(int m, int n, int k, float * A,  float* B,  float * out,  int lda, int ldb, int ldc, int bits);
template void gemm_host<half>(int m, int n, int k, half * A,  half* B,  half * out,  int lda, int ldb, int ldc, int bits);
template void extractOutliers<COL_TURING>(char * A, int *idx, char *out, int idx_size, int rows, int cols);
template void extractOutliers<COL_AMPERE>(char * A, int *idx, char *out, int idx_size, int rows, int cols);

template void spmm_coo_very_sparse_naive<half, 16>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, half *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);
template void spmm_coo_very_sparse_naive<signed char, 8>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, signed char *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);

template int igemmlt<32, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc, hipStream_t stream);
template int igemmlt<8, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc, hipStream_t stream);
template int igemmlt<8, 1>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc, hipStream_t stream);

template void transformRowToFormat<COL32, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL32, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 1>(char * A, char *out, int rows, int cols);

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 1, General8bit>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<half, 0, General8bit>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<half, 0, FP4>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<half, 0, NF4>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 1, General8bit>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 0, General8bit>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 0, FP4>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 0, NF4>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<__hip_bfloat16, 1, General8bit>(float * code, __hip_bfloat16 *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<__hip_bfloat16, 0, General8bit>(float * code, __hip_bfloat16 *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<__hip_bfloat16, 0, FP4>(float * code, __hip_bfloat16 *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<__hip_bfloat16, 0, NF4>(float * code, __hip_bfloat16 *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);

template void dequantizeBlockwise<float, General8bit>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<float, FP4>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<float, NF4>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<half, General8bit>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<half, FP4>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<half, NF4>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<__hip_bfloat16, General8bit>(float *code, unsigned char *A, float *absmax, __hip_bfloat16 *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<__hip_bfloat16, FP4>(float *code, unsigned char *A, float *absmax, __hip_bfloat16 *out, int blocksize, const int n, hipStream_t stream);
template void dequantizeBlockwise<__hip_bfloat16, NF4>(float *code, unsigned char *A, float *absmax, __hip_bfloat16 *out, int blocksize, const int n, hipStream_t stream);

#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float beta3, const float alpha, \
                const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(ADAM, __hip_bfloat16)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(MOMENTUM, __hip_bfloat16)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)
MAKE_optimizer32bit(RMSPROP, __hip_bfloat16)
MAKE_optimizer32bit(LION, half)
MAKE_optimizer32bit(LION, float)
MAKE_optimizer32bit(LION, __hip_bfloat16)
MAKE_optimizer32bit(ADAGRAD, half)
MAKE_optimizer32bit(ADAGRAD, float)
MAKE_optimizer32bit(ADAGRAD, __hip_bfloat16)
MAKE_optimizer32bit(ADEMAMIX, half)
MAKE_optimizer32bit(ADEMAMIX, __hip_bfloat16)
MAKE_optimizer32bit(ADEMAMIX, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)
MAKE_optimizerStatic8bit(LION, half)
MAKE_optimizerStatic8bit(LION, float)
MAKE_optimizerStatic8bit(ADAGRAD, half)
MAKE_optimizerStatic8bit(ADAGRAD, float)


#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float beta3, float alpha, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, bool skip_zeros, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(half, LION);
MAKE_optimizerStatic8bitBlockwise(float, LION);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, LION);
MAKE_optimizerStatic8bitBlockwise(half, ADAGRAD);
MAKE_optimizerStatic8bitBlockwise(float, ADAGRAD);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, ADAGRAD);
MAKE_optimizerStatic8bitBlockwise(half, ADEMAMIX);
MAKE_optimizerStatic8bitBlockwise(__hip_bfloat16, ADEMAMIX);
MAKE_optimizerStatic8bitBlockwise(float, ADEMAMIX);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);

template void transform<int8_t, ROW, COL, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL32, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, ROW, COL32, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_TURING, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_AMPERE, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, COL32, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, COL32, ROW, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);

template int get_leading_dim<ROW>(int dim1, int dim2);
template int get_leading_dim<COL>(int dim1, int dim2);
template int get_leading_dim<COL32>(int dim1, int dim2);
